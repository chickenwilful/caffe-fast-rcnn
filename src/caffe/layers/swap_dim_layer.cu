#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/swap_dim_layer.hpp"


namespace caffe {

template <typename Dtype>
__global__ void Forward(const int nthreads,
	const Dtype* bottom_data, const int num, int channels, int height, int width,
	Dtype *top_data) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		int w = index % width;
		int h = (index / width) % height;
		int c = (index / width / height) % channels;
		int n = index / width / height / channels;

		int bottom_index =  w * height * channels * num +  h * channels * num + c * num + n;
		top_data[index] = bottom_data[bottom_index]; 
	}
}


template <typename Dtype>
void SwapDimLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
			const vector<Blob<Dtype>*>& top) {
	const Dtype *bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();

	const int count = top[0]->count();
	const int num_ = top[0]->num();
	const int channels_ = top[0]->channels();
	const int height_ = top[0]->height();
	const int width_ = top[0]->width();
 
	Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, bottom_data, num_, channels_, height_, width_, top_data)
	
	CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void Backward(const int nthreads,
	const Dtype* top_diff, const int num, int channels, int height, int width,
	Dtype *bottom_diff) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		int w = index % width;
		int h = (index / width) % height;
		int c = (index / width / height) % channels;
		int n = index / width / height / channels;

		int bottom_index =  w * height * channels * num +  h * channels * num + c * num + n;
		bottom_diff[bottom_index] = top_diff[index]; 
	}
}


template <typename Dtype>
void SwapDimLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
			const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (!propagate_down[0]) {
		return;
	}

	const Dtype *top_diff = top[0]->gpu_diff();
	Dtype* bottom_Diff = bottom[0]->mutable_gpu_diff();
	const int count = top[0]->count();
	const int num_ = top[0]->num();
	const int channels = top[0]->channels();
	const int height_ = top[0]->height();
	const int width_ = top[0]->width();
	caffe_gpu_set(count, Dtype(0.), bottom_diff);


	Backward<Dtype><<<CAFFE_GET_BLOCKS(count)>>>, CAFFE_CUDA_NUM_THREADS>>>(
		count, top_diff, num_, channels_, height_, width_, bottom_diff);
	
	CUDA_POST_KERNEL_CHECK;	
}

INSTANTIATE_LAYER_GPU_FUNCS(SwapDimLayer);
} // namespace caffe
